
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define THREADS_PER_BLOCK 1024
#define MAX_NUMBER_BLOCKS 2496

/*******************************************************
*               RUNTIME MEASURING METHODS              *
*******************************************************/
struct timeval start, end; 

void starttime(){
	gettimeofday(&start,0);
}

void endtime(const char* c){
	gettimeofday(&end, 0);
	double elapsed = (end.tv_sec - start.tv_sec) * 1000.0 + (end.tv_usec - start.tv_usec) / 1000.0;
	printf("%s: %f ms\n", c, elapsed);
}

/******************************************************
*   	           CUDA METHODS                       *
******************************************************/

__global__ void validSets(int* fTable, int cardinality, int nCr, int mSupport){
	int tIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if((tIndex < (cardinality + 1) * nCr) && (tIndex % (cardinality + 1) == cardinality)){
		if(fTable[tIndex] < mSupport){
			fTable[tIndex] = 0;
		}
	}
}

__global__ void counting(int* fTable, char* tTable, int row, int col, int nCr, int cardinality){
	
	__shared__ int cache[THREADS_PER_BLOCK]; //cache memory that is shared by all the threads within a block
	int bIndex = blockIdx.x; //the index value of the core
	int cacheIndex = threadIdx.x; //each thread within a core has a corresponding cache index where it stores its values

	//enter a block loop where the core index must remain lower than the amount of item sets present in the frequency table
	//at the end of each iteration the core index is increased by the amount of cores being used and loops again if possible
	for(int h = bIndex; h < nCr; h+= gridDim.x){
		
		int tIndex = threadIdx.x; //the index value of the individual thread
		int sum = 0; //keeps track of how many times an item set has been found
		int found; //a boolean value that indicates whether an item set is present within a transaction; either 0 or 1

		//enter a thread loop where i represents which transaction being scanned. Each thread within a core scans a
		// different transaction; the loop is necessary since there aren't enough threads for each transaction. Whenever
		// a scan is done i is incremented by th number of threads per block
		for(int i = tIndex; i < row; i+= blockDim.x){

			found = 1;

			//enter a loop where j represents the specific item within an item set; the iterations within the for loop
			// is dependent on the cardinality of the item sets
			for(int j = 0; j < cardinality; j++){
				
				//if an item indicated in the frequency table is not found in the transaction found is set to 0; i.e. false
				if(tTable[i * col + (fTable[bIndex * (cardinality + 1) + j])] != '1'){
					found = 0;
				}
			}	

			//if found equals 1 then the sum variable is incremented by 1
			if(found == 1){
				sum++;
			}	
		}
		
		//once any given thread exits the thread the thread loop it stores its sum value to its corresponding cache index 
		cache[cacheIndex] = sum;
		
		//the threads are synced before the overall sum is calculated to ensure all threads have finished counting;
		__syncthreads();

		//the cache is then reduced to obtain the total sum for any given item set every iteration adds two cache location 
		//together until the sum is stored at cache[0]
		int k = THREADS_PER_BLOCK/2;
		while(k != 0){
			if(cacheIndex < k){
				cache[cacheIndex] += cache[cacheIndex + k];
			}
			__syncthreads();
			k /= 2;
		}

		//takes the overall of the item set for the core index that is monitoring this specific item set and enters it into the 
		//corresponding count column within the frequency table
		if(cacheIndex == 0){
			fTable[bIndex * (cardinality + 1) + cardinality] = cache[0];
		}
		__syncthreads();
		//the core index value is incremented by the number of cores being used
		bIndex += gridDim.x;
	}
}


//factorial function
long int factorial(int x){
	int count = x;
	while (count > 1){
		x = x * (count - 1);
		count--;
	}
	if(x == 0){
		x = 1;
	}	
	return x;	
}

//combinatorics function
long int nCr(int n, int r){
	int y;
	int z;
	int w = n - 1;
	int init = n;
	int x;
	if(r > (n-r)){
	y = r;	
	}
	else{
		y = (n-r);
	}

	z = n - y;
	while(z > 1){
		n = n * w;
		w--;
		z--;
	}
	if( r > (init - r)){
		x = n/factorial(init - r);
	}
	else{
		x = n/factorial(r);
	}
	
	return  x;

}

int main() {
	
	/************************************************************************************
	*                                  Variable Declarations                            *
	************************************************************************************/	
	FILE *fPointer;
	int max = 0; 
	int size = 0; //Contains the number of lines in the given database
	int cardinality = 1; //Contains the initial cardinality of the item sets
    	int temp;
	int i = 0;
	int j, k, num, count;
	int mSupport = 8000; //Contains the support count; set to approx 10% of all transactions
	char val;
	int numBlocks = 0; 
	//While loop that traverses through the database and returns the number of transactions  
	fPointer = fopen("retail.dat", "r"); 
   	fscanf(fPointer, "%c", &val);
   	while(!feof(fPointer)){
        	if(val == '\n'){
            		size++;
        	}
       		fscanf(fPointer, "%c", &val);
    	}
    	fclose(fPointer);

    	fPointer = fopen("retail.dat", "r");
   	fscanf(fPointer, "%d", &temp);
	
	//Traverses through each transaction in order to find the max value.
    	while(!feof(fPointer)){
        	fscanf(fPointer, "%d", &temp);
        	if(max < temp){
            		max = temp;
        	}
    	}	
   	fclose(fPointer);

	printf("DATA FILE PARSED\n");
	printf("============================================\n");	
	printf("Total number of transactions found: %d\n", size);
	printf("Maximum number of unique items found: %d\n", max+1);
	printf("============================================\n");	
	printf("APRIORI IMPLEMENTATION BEGINS\n");

	starttime();

	//Creation of table
	char *cTable = (char*)malloc(sizeof(char) * (max + 1) * size); //Allocates an array of characters for each transaction	
	
	for(i=0; i < (max+1)*size; i++) {
	//	memset(cTable[i], '\0', sizeof(char) * (max + 1) * size); //Initialize all values to 0.
		cTable[i] = '\0';
	}

    	char line[400];
    	char *cNum;
    	fPointer = fopen("retail.dat", "r");
	for(i = 0; i < size; i++){
		fgets(line, 400, fPointer);

        	cNum = strtok(line, " \n");
        	
		while(cNum != NULL){
            		num = atoi(cNum);
            		cTable[i * (max + 1) + num] = '1';
            		cNum = strtok(NULL, " \n");
        	}	
    	}

	//Creating copy of transaction table in the video card memmory
	char* gpuT;
	hipMalloc(&gpuT, size * (max + 1) * sizeof(char));
	hipMemcpy(gpuT, cTable, (size * (max + 1) * sizeof(char)), hipMemcpyHostToDevice);

	//Creates a frequency table of item sets with a Cardinality of 1; where the array index represents the item 
	//number. All the items have their counts initially set to zero
	int * fTable = (int *)malloc((max + 1) * (cardinality + 1) * sizeof(int));
	for(i = 0; i < max + 1; i++){
		fTable[i * (cardinality + 1)] = i;
		fTable[(i * (cardinality + 1)) + cardinality] = 0;
	}

	int* gpuF;
	hipMalloc(&gpuF, (max + 1) * (cardinality + 1) * sizeof(int));
	hipMemcpy(gpuF, fTable, (max + 1) * (cardinality + 1) * sizeof(int), hipMemcpyHostToDevice);

	//setting the number of cores to be used by the gpu
	numBlocks = (max + 1);
	if(numBlocks > MAX_NUMBER_BLOCKS){
		numBlocks = MAX_NUMBER_BLOCKS;
	}
	counting<<< numBlocks, THREADS_PER_BLOCK>>>(gpuF, gpuT, size, (max + 1), (max + 1),  cardinality);
	
	//setting the number of cores to be used by the gpu
	numBlocks = (max + 1) * (cardinality + 1)/ THREADS_PER_BLOCK + 1;
	if(numBlocks > MAX_NUMBER_BLOCKS){
		numBlocks = MAX_NUMBER_BLOCKS;
	}
	validSets<<< numBlocks, THREADS_PER_BLOCK>>>(gpuF, cardinality, max + 1, mSupport);
	hipMemcpy(fTable, gpuF, ((max + 1) * (cardinality + 1) * sizeof(int)), hipMemcpyDeviceToHost);
	hipFree(gpuF);

	//invalidating elements that are below the support count and counting the remaining eligible elements
	count = 0;
	for(i = 0; i < (max + 1); i++){
		if (fTable[i * (cardinality + 1) + cardinality] != 0){
			count++;
		}
	}

	//creating new table consisting of only valid items
        int iTable[count];
        j = 0;
        for(i = 0; i < (max + 1); i++){
                if (fTable[i * (cardinality + 1) + cardinality] != 0){
                        iTable[j] = fTable[i * (cardinality + 1)];			
                        j++;
                }
        }

	//creating a tabel to hold the current valid items item and their the a variable for the count of the count
	int * vTable = iTable;
	int lastCount = count;

	while(count > 1){
		cardinality++;

		//temporary array that will hold the new item sets		
		int temp[nCr(count, cardinality) * (cardinality + 1)];

		//array of previous items sets
		int oldSets[nCr(lastCount, cardinality - 1) * cardinality];

		//array that hold one old item set for insertion into table
		int oldEntry[cardinality - 1];

                //function populates old  item set
                k = 0;
                if(cardinality - 1 <= lastCount){
                        for(i = 0; (oldEntry[i] = i) < cardinality - 2; i++); 
                        for(i = 0; i < cardinality - 1; i++){
                                oldSets[(k * cardinality) + i] = vTable[oldEntry[i]];
                        }
                        k++;
                        for(;;){
                                for( i = cardinality - 2; i >= 0 && oldEntry[i] == (lastCount - (cardinality - 1) + i); i--);
                                if(i < 0){
                                        break;
                                }
                                else{
                                        oldEntry[i]++;
                                        for(++i; i < cardinality - 1; i++){
                                                oldEntry[i] = oldEntry[i - 1] + 1;
                                        }
                                        for(j = 0; j < cardinality - 1; j++){
                                                oldSets[(k * cardinality) + j] = vTable[oldEntry[j]];
                                        }
                                        k++;
                                }
                        }
                }

                for(i = 0; i < nCr(lastCount, cardinality - 1); i++){
                        oldSets[(i * cardinality) + cardinality - 1] = 0;
                }

		//array that will hold the information for a single item set before it is added to the 
		//array of all item sets
		int entry[cardinality];

		//function populates new item set
		k = 0;
		if(cardinality <= count){
			for(i = 0; (entry[i] = i) < cardinality - 1; i++);			
			for(i = 0; i < cardinality; i++){
				temp[(k*(cardinality + 1)) + i] = vTable[entry[i]];
			}
			k++;
			for(;;){
				for( i = cardinality - 1; i >= 0 && entry[i] == (count - cardinality + i); i--);
				if(i < 0){
					break;
				}
				else{
					entry[i]++;
					for(++i; i < cardinality; i++){
						entry[i] = entry[i - 1] + 1;
					}
					for(j = 0; j < cardinality; j++){
						temp[(k*(cardinality + 1)) + j] = vTable[entry[j]];
					}
					k++;
				}
			}
		}


						      
		for(i = 0; i < nCr(count, cardinality); i++){
			temp[(i*(cardinality + 1)) + cardinality ] = 0;
		}

		//counting the amount of instances of the item sets amongst the transactions
		int * gpuSet;
		hipMalloc(&gpuSet, sizeof(int) * (cardinality + 1) * nCr(count, cardinality));
		hipMemcpy(gpuSet, temp, sizeof(int) * (cardinality + 1) * nCr(count, cardinality), hipMemcpyHostToDevice);
		numBlocks = nCr(count, cardinality);
		if(numBlocks > MAX_NUMBER_BLOCKS){
			numBlocks = MAX_NUMBER_BLOCKS;
		}
		counting<<< numBlocks, THREADS_PER_BLOCK>>>(gpuSet, gpuT, size, max + 1, nCr(count, cardinality), cardinality);
		hipMemcpy(temp, gpuSet, sizeof(int) * (cardinality + 1) * nCr(count, cardinality), hipMemcpyDeviceToHost);
		hipFree(gpuSet);
		
                //counting the amount of instances of the item sets amongst the transactions
		hipMalloc(&gpuSet, sizeof(int) * cardinality * nCr(lastCount, cardinality - 1));
		hipMemcpy(gpuSet, oldSets, sizeof(int) * cardinality * nCr(lastCount, cardinality - 1), hipMemcpyHostToDevice);
		numBlocks = nCr(lastCount, cardinality - 1);
		if(numBlocks > MAX_NUMBER_BLOCKS){
			numBlocks = MAX_NUMBER_BLOCKS;
		}
		counting<<< numBlocks, THREADS_PER_BLOCK>>>(gpuSet, gpuT, size, max + 1, nCr(lastCount, cardinality - 1), cardinality - 1);
		hipMemcpy(oldSets, gpuSet, sizeof(int) * cardinality * nCr(lastCount, cardinality - 1), hipMemcpyDeviceToHost);
		hipFree(gpuSet);

		//invalidating elements that are below the support count and counting the remaining eligible elements
        	int tCount = count;
		lastCount = tCount;
		count = 0;
        	for(i = 0; i < nCr(tCount, cardinality); i++){
                	if (temp[(i*(cardinality + 1)) + cardinality] < mSupport){
                        	temp[(i * (cardinality + 1)) + cardinality] = 0;
                	}	
                	else{
                        	count++;
                	}
        	}		

		//set Table of valid items
		char valid[max + 1];
		for(i = 0; i <= max; i++){
			valid[i] = '\0';
		}

		for(i = 0; i < nCr(tCount, cardinality); i++){
			for(j = 0; j < cardinality; j++){
				if(temp[(i * (cardinality + 1)) + cardinality] > 0){
					valid[temp[(i * (cardinality + 1)) + j]] = '1';
				}
			}
		}

        	//creating new table consisting of only valid items
        	int rTable[count];
		count = 0;
        	j = 0;
        	for(i = 0; i <= max; i++){
                	if (valid[i] == '1'){
                        	rTable[j] = i;
                        	j++;
				count++;
	                }
        	}	
		vTable = rTable;

		if(count == 0){
			printf("\n=============== MOST FREQUENT SUBSETS ================\n");
	   
	        	for(i = 0; i < nCr(lastCount, cardinality - 1); i++){
				if(oldSets[(i * cardinality) + (cardinality-1)] > mSupport){
                                        printf("Set: {");
                                }
               			for(j = 0; j < cardinality; j++){
					if(oldSets[(i * cardinality) + (cardinality-1)] > mSupport){
                               			if(j == cardinality - 1){
							printf("}\t\tCount: %d\n", oldSets[(i * cardinality) + j]);
						}
						else{
							printf("'%d'", oldSets[(i * cardinality) + j]);
						}
                       		 	}	
               		 	}        
			}
			printf("\n");	
		}
	}

	endtime("Total Parallelized Implementation Time" );
}

